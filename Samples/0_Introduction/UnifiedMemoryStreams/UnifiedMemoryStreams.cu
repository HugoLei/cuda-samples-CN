#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a simple task consumer using threads and streams
 * with all data in Unified Memory, and tasks consumed by both host and device
 */

/*
* 此示例使用线程和流实现了一个简单的任务消费者
* 所有数据都在统一内存中，任务由主机和设备共同消耗
*/

/*
* Unified Memory介绍
* cuda unified memory是一种在CUDA编程模型中使用的内存管理机制。
* 它允许开发者以一种统一的方式访问和管理CPU和GPU的内存，从而简化了编程的复杂性。
* 在Unified Memory模型中，CUDA会自动管理数据在CPU和GPU之间的迁移，使得数据能够在需要的时候出现在正确的位置上。
* 这种机制消除了手动管理数据传输的需要，使得代码更加简洁和易于维护。
* 此外，Unified Memory还提供了内存超额订阅功能，即可以分配超过物理GPU内存大小的内存空间，CUDA会在需要时自动进行数据的换入换出。
*/

// system includes
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#ifdef USE_PTHREADS
#include <pthread.h>
#else
#include <omp.h>
#endif
#include <stdlib.h>

// cuBLAS
#include <hipblas.h>

// utilities
#include <hip/hip_runtime_api.h>

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
// SRAND48 and DRAND48 don't exist on windows, but these are the equivalent
// functions
void srand48(long seed) { srand((unsigned int)seed); }
double drand48() { return double(rand()) / RAND_MAX; }
#endif

const char *sSDKname = "UnifiedMemoryStreams";

// simple task
template <typename T>
struct Task {
  unsigned int size, id;
  T *data;
  T *result;
  T *vector;

  Task() : size(0), id(0), data(NULL), result(NULL), vector(NULL){};
  Task(unsigned int s) : size(s), id(0), data(NULL), result(NULL) {
    // allocate unified memory -- the operation performed in this example will
    // be a DGEMV
    // 这个例子中的逻辑是执行DGEMV
    // DGEMV 密集型通用矩阵-向量乘法（Dense General Matrix-Vector Multiplication）

    /* 
    * hipMallocManaged函数介绍
    * 是 CUDA 编程中的一个重要函数，它用于在 CUDA 程序中分配一块可由 CPU 和 GPU 共同访问和管理的内存。
    * 这种内存称为托管内存（managed memory）。
    * 使用托管内存可以大大简化数据在 CPU 和 GPU 之间的传输过程，因为 CUDA 运行时会自动处理这些内存的数据迁移，从而减少了程序员的负担。
    */
    checkCudaErrors(hipMallocManaged(&data, sizeof(T) * size * size));
    checkCudaErrors(hipMallocManaged(&result, sizeof(T) * size));
    checkCudaErrors(hipMallocManaged(&vector, sizeof(T) * size));

    /*
    * hipDeviceSynchronize函数介绍
    * 是CUDA Runtime API中的一个函数，用于等待直到所有之前提交的CUDA命令都完成。
    * 这包括在调用hipDeviceSynchronize之前的所有CUDA内核启动、内存复制和内存分配操作等。
    */
    checkCudaErrors(hipDeviceSynchronize());
  }

  ~Task() {
    // ensure all memory is deallocated
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(data));
    checkCudaErrors(hipFree(result));
    checkCudaErrors(hipFree(vector));
  }

  // 若是通过无参构造函数Task()构造的Task，可以继续调用allocate来分配unified memory
  void allocate(const unsigned int s, const unsigned int unique_id) {
    // allocate unified memory outside of constructor
    id = unique_id;
    size = s;
    checkCudaErrors(hipMallocManaged(&data, sizeof(T) * size * size));
    checkCudaErrors(hipMallocManaged(&result, sizeof(T) * size));
    checkCudaErrors(hipMallocManaged(&vector, sizeof(T) * size));
    checkCudaErrors(hipDeviceSynchronize());

    // populate data with random elements
    // 数据初始化，随机值
    for (unsigned int i = 0; i < size * size; i++) {
      data[i] = drand48();
    }

    for (unsigned int i = 0; i < size; i++) {
      result[i] = 0.;
      vector[i] = drand48();
    }
  }
};

#ifdef USE_PTHREADS
struct threadData_t {
  int tid;
  Task<double> *TaskListPtr;
  hipStream_t *streams;
  hipblasHandle_t *handles;
  int taskSize;
};

typedef struct threadData_t threadData;
#endif

// simple host dgemv: assume data is in row-major format and square
template <typename T>
// 在CPU上执行dgemv，假设数据是行优先存储，且矩阵是个方阵
/*
* [row-major format]介绍
* 概述：将多维数组打平成一维数组
* 在计算机科学中，多维数组（如矩阵）可以以行优先或列优先的方式存储在内存中。
* 在行优先格式中，数组的元素首先按行填充，即首先填充第一行的所有元素，然后是第二行，依此类推。（此处举例是二维矩阵，按行打平）
*/
void gemv(int m, int n, T alpha, T *A, T *x, T beta, T *result) {
  // rows
  for (int i = 0; i < n; i++) {
    result[i] *= beta;

    for (int j = 0; j < n; j++) {
      result[i] += A[i * n + j] * x[j];
    }
  }
}

// execute a single task on either host or device depending on size
// 执行一个task，根据task的size决定是在CPU还是在GPU上运行
// size < 100 在CPU上执行，否则在GPU上执行
#ifdef USE_PTHREADS
void *execute(void *inpArgs) {
  threadData *dataPtr = (threadData *)inpArgs;
  hipStream_t *stream = dataPtr->streams;
  hipblasHandle_t *handle = dataPtr->handles;
  int tid = dataPtr->tid;

  for (int i = 0; i < dataPtr->taskSize; i++) {
    Task<double> &t = dataPtr->TaskListPtr[i];

    if (t.size < 100) {
      // perform on host
      printf("Task [%d], thread [%d] executing on host (%d)\n", t.id, tid,
             t.size);

      // attach managed memory to a (dummy) stream to allow host access while
      // the device is running
      checkCudaErrors(
          hipStreamAttachMemAsync(stream[0], t.data, 0, hipMemAttachHost));
      checkCudaErrors(
          hipStreamAttachMemAsync(stream[0], t.vector, 0, hipMemAttachHost));
      checkCudaErrors(
          hipStreamAttachMemAsync(stream[0], t.result, 0, hipMemAttachHost));
      // necessary to ensure Async hipStreamAttachMemAsync calls have finished
      checkCudaErrors(hipStreamSynchronize(stream[0]));
      // call the host operation
      gemv(t.size, t.size, 1.0, t.data, t.vector, 0.0, t.result);
    } else {
      // perform on device
      printf("Task [%d], thread [%d] executing on device (%d)\n", t.id, tid,
             t.size);
      double one = 1.0;
      double zero = 0.0;

      // attach managed memory to my stream
      checkCudaErrors(hipblasSetStream(handle[tid + 1], stream[tid + 1]));
      checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.data, 0,
                                               hipMemAttachSingle));
      checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.vector, 0,
                                               hipMemAttachSingle));
      checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.result, 0,
                                               hipMemAttachSingle));
      // call the device operation
      checkCudaErrors(hipblasDgemv(handle[tid + 1], HIPBLAS_OP_N, t.size, t.size,
                                  &one, t.data, t.size, t.vector, 1, &zero,
                                  t.result, 1));
    }
  }

  pthread_exit(NULL);
}
#else
template <typename T>
void execute(Task<T> &t, hipblasHandle_t *handle, hipStream_t *stream,
             int tid) {
  if (t.size < 100) {
    // perform on host
    printf("Task [%d], thread [%d] executing on host (%d)\n", t.id, tid,
           t.size);

    // attach managed memory to a (dummy) stream to allow host access while the
    // device is running
    checkCudaErrors(
        hipStreamAttachMemAsync(stream[0], t.data, 0, hipMemAttachHost));
    checkCudaErrors(
        hipStreamAttachMemAsync(stream[0], t.vector, 0, hipMemAttachHost));
    checkCudaErrors(
        hipStreamAttachMemAsync(stream[0], t.result, 0, hipMemAttachHost));
    // necessary to ensure Async hipStreamAttachMemAsync calls have finished
    checkCudaErrors(hipStreamSynchronize(stream[0]));
    // call the host operation
    gemv(t.size, t.size, 1.0, t.data, t.vector, 0.0, t.result);
  } else {
    // perform on device
    printf("Task [%d], thread [%d] executing on device (%d)\n", t.id, tid,
           t.size);
    double one = 1.0;
    double zero = 0.0;

    // attach managed memory to my stream
    checkCudaErrors(hipblasSetStream(handle[tid + 1], stream[tid + 1]));
    checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.data, 0,
                                             hipMemAttachSingle));
    checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.vector, 0,
                                             hipMemAttachSingle));
    checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.result, 0,
                                             hipMemAttachSingle));
    // call the device operation
    checkCudaErrors(hipblasDgemv(handle[tid + 1], HIPBLAS_OP_N, t.size, t.size,
                                &one, t.data, t.size, t.vector, 1, &zero,
                                t.result, 1));
  }
}
#endif

// populate a list of tasks with random sizes
// 初始化Task，其size被初始化为随机数
template <typename T>
void initialise_tasks(std::vector<Task<T> > &TaskList) {
  for (unsigned int i = 0; i < TaskList.size(); i++) {
    // generate random size
    int size;
    size = std::max((int)(drand48() * 1000.0), 64);
    TaskList[i].allocate(size, i);
  }
}

int main(int argc, char **argv) {
  // set device
  hipDeviceProp_t device_prop;
  int dev_id = findCudaDevice(argc, (const char **)argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

  if (!device_prop.managedMemory) {
    // This samples requires being run on a device that supports Unified Memory
    fprintf(stderr, "Unified Memory not supported on this device\n");

    exit(EXIT_WAIVED);
  }

  if (device_prop.computeMode == hipComputeModeProhibited) {
    // This sample requires being run with a default or process exclusive mode
    fprintf(stderr,
            "This sample requires a device in either default or process "
            "exclusive mode\n");

    exit(EXIT_WAIVED);
  }

  // randomise task sizes
  int seed = (int)time(NULL);
  srand48(seed);

  // set number of threads
  const int nthreads = 4;

  // number of streams = number of threads
  /*
  * [stream]介绍
  * CUDA的stream（流）是一种处理并行操作的方式，它允许开发者在CUDA程序中组织和管理并发执行的任务。
  * 通过使用不同的流，可以并行执行多个核函数（kernels）或内存传输操作，从而实现更高的硬件利用率和程序性能。
  * 此处创建了streams数组，可以让多个thread在不同的stream中并发执行（并发的内存操作，并发的kernel执行）
  */
  hipStream_t *streams = new hipStream_t[nthreads + 1];
  hipblasHandle_t *handles = new hipblasHandle_t[nthreads + 1];

  for (int i = 0; i < nthreads + 1; i++) {
    checkCudaErrors(hipStreamCreate(&streams[i]));
    checkCudaErrors(hipblasCreate(&handles[i]));
  }

  // create list of N tasks
  unsigned int N = 40;
  std::vector<Task<double> > TaskList(N);
  initialise_tasks(TaskList);

  printf("Executing tasks on host / device\n");

// run through all tasks using threads and streams
#ifdef USE_PTHREADS
  pthread_t threads[nthreads];
  threadData *InputToThreads = new threadData[nthreads];

  for (int i = 0; i < nthreads; i++) {
    checkCudaErrors(hipSetDevice(dev_id));
    InputToThreads[i].tid = i;
    InputToThreads[i].streams = streams;
    InputToThreads[i].handles = handles;

    if ((TaskList.size() / nthreads) == 0) {
      InputToThreads[i].taskSize = (TaskList.size() / nthreads);
      InputToThreads[i].TaskListPtr =
          &TaskList[i * (TaskList.size() / nthreads)];
    } else {
      if (i == nthreads - 1) {
        InputToThreads[i].taskSize =
            (TaskList.size() / nthreads) + (TaskList.size() % nthreads);
        InputToThreads[i].TaskListPtr =
            &TaskList[i * (TaskList.size() / nthreads) +
                      (TaskList.size() % nthreads)];
      } else {
        InputToThreads[i].taskSize = (TaskList.size() / nthreads);
        InputToThreads[i].TaskListPtr =
            &TaskList[i * (TaskList.size() / nthreads)];
      }
    }

    pthread_create(&threads[i], NULL, &execute, &InputToThreads[i]);
  }
  for (int i = 0; i < nthreads; i++) {
    pthread_join(threads[i], NULL);
  }
#else
  omp_set_num_threads(nthreads);
#pragma omp parallel for schedule(dynamic)
  for (int i = 0; i < TaskList.size(); i++) {
    checkCudaErrors(hipSetDevice(dev_id));
    int tid = omp_get_thread_num();
    execute(TaskList[i], handles, streams, tid);
  }
#endif

  hipDeviceSynchronize();

  // Destroy CUDA Streams, cuBlas handles
  for (int i = 0; i < nthreads + 1; i++) {
    hipStreamDestroy(streams[i]);
    hipblasDestroy(handles[i]);
  }

  // Free TaskList
  std::vector<Task<double> >().swap(TaskList);

  printf("All Done!\n");
  exit(EXIT_SUCCESS);
}
